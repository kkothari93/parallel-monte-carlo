#include "hip/hip_runtime.h"
#include <ctime>
#include "hip/hip_runtime.h"
#include "math.h"
#include <stdio.h>
#include <stdlib.h>
#include <>
#include <hip/device_functions.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <string.h>
#include "device_atomic_functions.h"


#define N_ATOMS 1331
#define L 11.6328f
#define beta 0.3
#define cellsPerSide 4
#define w 3
#define nmax 35
#define BLOCK_SIZE 1024
#define n_M 40
#define simga 0.5
#define dimCB 8

#include "subsweep.h"

const int CPS2 = cellsPerSide*cellsPerSide;
const int CPS3 = CPS2*cellsPerSide;



// Fisher Yates shuffle as described on wiki
void FY_Shuffle(int * a, int n){
	int i, randindex;
	int temp;
	srand(time(NULL));
	for (i = n - 1; i > 0; i--){
		randindex = rand() % (i + 1);
		temp = a[n - i];
		a[n - i] = a[randindex];
		a[randindex] = temp;
	}
}


__global__ void init_r(float* r, int N_cube){
	int ix = threadIdx.x + blockDim.x* blockIdx.x;
	int iy = threadIdx.y + blockDim.y* blockIdx.y;
	int iz = threadIdx.z + blockDim.z* blockIdx.z;
	if (ix < N_cube & iy < N_cube & iz < N_cube){
		int index = ix + iy*N_cube + iz * N_cube * N_cube;
		//if (index == 1001){ printf("Init r kernel success! "); }
		r[index] = L / 2.0 * (1.0 - float(2 * ix + 1) / N_cube);
		r[index + N_ATOMS] = L / 2.0 * (1.0 - float(2 * iy + 1) / N_cube);
		r[index + 2 * N_ATOMS] = L / 2.0 * (1.0 - float(2 * iz + 1) / N_cube);
	}
}

//void assign(float *r, float *disk,int* index, int* n, int* presum){
//	int i, cellx, celly, cellz;
//	int write_index;
//	presum[0] = 0;
//	for (i = 0; i < CPS3; i++){
//		n[i] = 0;
//	}
//	for (i = 0; i < N_ATOMS; i++){
//		cellx = int((r[i] + L / 2.0f) / L*cellsPerSide);
//		celly = int((r[i + N_ATOMS] + L / 2.0f) / L*cellsPerSide);
//		cellz = int((r[i + 2 * N_ATOMS] + L / 2.0f) / L*cellsPerSide);
//		write_index = cellx + celly*cellsPerSide + cellz*CPS2;
//		index[i] = write_index;
//		n[write_index] += 1;
//		
//	}
//	//for (i = 1; i < CPS3; i++){
//	//	presum[i] = n[i] + presum[i - 1];
//	//}
//	//for (i = 0; i < N_ATOMS; i++){
//	//	disk[presum[index[i]]] = r[i];
//	//	disk[presum[index[i]] + n[index[i]]] = r[i+N_ATOMS];
//	//	disk[presum[index[i]] + 2 * n[index[i]]] = r[i + 2 * N_ATOMS];
//	//}
//}

// assign kernel - takes the positions and assigns them a cell index
__global__ void assign(float*r, float* disk, short int*n){
	__shared__ float r_sh[N_ATOMS*3];
	__shared__ short int n_sh[CPS3];
	int tid = threadIdx.x + blockIdx.x*blockDim.x;
	float xub, xlb, yub, ylb, zub, zlb, x, y, z;
	int cellx,celly,cellz, i;
	for (i = 0; i < int(ceil(3 * N_ATOMS / (float)BLOCK_SIZE)); i++){
		if (i*BLOCK_SIZE + tid < 3 * N_ATOMS){
			// storing positions in shared memory for faster access
			// will have to change this if N_ATOMS > 2048
			r_sh[i*BLOCK_SIZE + tid] = r[i*BLOCK_SIZE + tid];
		}
	}
	__syncthreads();
	if (tid < CPS3){
		n_sh[tid] = 0;
	}
	__syncthreads(); //check for syncthreads();

	// Approach 1 - with atomic add
	/****************************************/
	/*     cannot avoid race condition      */
	/****************************************/
	//if (tid < N_ATOMS){
	//	cellx = int((r_sh[tid] + L / 2.0f) / L*cellsPerSide);
	//	celly = int((r_sh[tid + N_ATOMS] + L / 2.0f) / L*cellsPerSide);
	//	cellz = int((r_sh[tid + 2 * N_ATOMS] + L / 2.0f) / L*cellsPerSide);
	//	int write_index = cellx + celly * cellsPerSide + cellz * CPS2;
	//	disk[write_index*nmax * 3 + n_sh[write_index]] = r_sh[tid];
	//	disk[write_index*nmax * 3 + n_sh[write_index]+ nmax] = r_sh[tid];
	//	disk[write_index*nmax * 3 + n_sh[write_index] + 2 *nmax] = r_sh[tid];
	//	atomicAdd(&n_sh[write_index], 1);
	//}

	// Approach 2 - no atomic add
	/****************************************/
	/*   No race - but redundancy in checks */
	/****************************************/
	if (tid < CPS3){
		cellz = tid / CPS2;
		celly = (tid - cellz*CPS2) / cellsPerSide;
		cellx = tid - cellz*CPS2 - celly * cellsPerSide;
		xlb = cellx*w - L / 2.0f; xub = xlb + w;
		ylb = celly*w - L / 2.0f; yub = ylb + w;
		zlb = cellz*w - L / 2.0f; zub = zlb + w;
		for (i = 0; i < N_ATOMS; i++){
			x = r_sh[i];  y = r_sh[i + N_ATOMS]; z = r_sh[i + 2 * N_ATOMS];
			if ((x <= xub && x > xlb) && (y <= yub &&  y > ylb) && (z <= zub && z > zlb)){
				disk[tid*nmax * 3 + n_sh[tid]] = x;
				disk[tid*nmax * 3 + n_sh[tid] + nmax] = y;
				disk[tid*nmax * 3 + n_sh[tid] + nmax * 2] = z;
				n_sh[tid] += 1;
			}
		}
	}
	//if (tid == 0) { printf("cell 0 has %i particles\n", n_sh[0] ); }
	if (tid < CPS3){
		n[tid] = n_sh[tid];
	}
}

// Sub-sweep kernel

// cell redraw boundaries kernel


int main(){
	// test input to check Fisher yates
	//float a[8] = { 1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0 };
	//int len = 8;
	//FY_Shuffle(&a[0], len);
	//for (int i = 0; i < len; i++){
	//	printf("%f \t", a[i]);
	//}

	// declare variables
	float * r;
	float* d_r;
	float * disk;
	float * d_disk;
	float * disk_dbl;
	float * n_dbl;
	short int * n;
	short int * d_n;
	hipError_t state;

	int rsize = 3 * N_ATOMS * sizeof(float);
	int nsize = sizeof(short int) * CPS3;
	int disksize = sizeof(float) * 3 * nmax * CPS3;
    	int cboard_index[8]= {0,1,2,3,4,5,6,7};
    	int ii, i;
	int f;
	float d;

	// allocate space on CPU
	r = (float *)malloc(rsize);
	disk = (float *)malloc(disksize);
	n = (short int *)malloc(nsize);

	// allocate space on GPU
	hipMalloc((void **)&d_r, rsize);
	hipMalloc((void **)&d_disk, disksize);
	hipMalloc((void **)&d_n, nsize);
	hipMalloc((void **)&disk_dbl, disksize);
	hipMalloc((void **)&n_dbl, nsize);

	// initialize positions
	int N_cube = int(cbrt(float(N_ATOMS)));
	int gd = int(ceil(double(N_ATOMS) / N_cube));
	dim3 gridSize(gd, gd, gd);
	dim3 blockSize(10, 10, 10);
	init_r << <gridSize, blockSize >> >(d_r, N_cube);
	state = hipDeviceSynchronize();
	if (state != hipSuccess){
		printf("Init r kernel failed : ", hipGetErrorString(state));
	}

	// Check positions generated on GPU
	//hipMemcpy(r, d_r, rsize, hipMemcpyDeviceToHost);
	//for (int ii = 0; ii < N_ATOMS * 3; ii++){
	//printf("%f \t", r[ii]);
	//if ((ii + 1) % N_ATOMS == 0)
	//printf("\n");
	//}

	// cell assignment
	assign << <int(ceil(float(CPS3)/BLOCK_SIZE)), BLOCK_SIZE >> >(d_r, d_disk, d_n);
	state = hipDeviceSynchronize();
	if (state != hipSuccess){
		printf("Assign kernel failed : ", hipGetErrorString(state));
	}
	//check assignment kernel
	short int total = 0;
	//hipMemcpy(disk, d_disk, disksize, hipMemcpyDeviceToHost);
	//hipMemcpy(n, d_n, nsize, hipMemcpyDeviceToHost);
	//for (ii = 0; ii < nsize / sizeof(short int); ii++){
	//	printf("# particles in cell %i : %i\n", ii, n[ii]);
	//	total += n[ii];
		/*for (int j = 0; j < n[ii]; j++){
			printf("Particle %i at : %f %f %f\n", j, disk[j + ii*nmax * 3], disk[j + ii*nmax * 3 + nmax], disk[j + ii*nmax * 3 + 2*nmax]);
		}*/
	}
	//if (total != N_ATOMS){ printf("Sanity check failed! All atoms may not be assigned to grids.\nPlease uncomment the appropriate code for sanity check to run or check the assign kernel parameters\n"); }
	//else{ printf("Sanity check passed!\n"); }
    	FY_Shuffle(cboard_index, dimCB);
    	for(i = 0; i < dimCB; i++){
        	r = itoa(cboard_index[i],2);
		// sub-sweep kernel
    		subsweep_kernel<<<int(ceil(float(CPS3)/BLOCK_SIZE)), CPS3>>>(d_disk,d_n,r);
    		state = hipDeviceSynchronize();
    		if (state != hipSuccess){
        		printf("Subsweep failed : ", hipGetErrorString(state));
    		}
		f = rand()%3 - 1;
		d = float (rand())/RAND_MAX * w - w/2.0f;
		// cell redraw boundaries kernel
    		shiftCells<<<int(ceil(float(CPS3)/BLOCK_SIZE)), CPS3>>>(d_disk, d_n, f, d, disk_dbl, n_dbl);
		// memcpy results to CPU
		hipMemcpy(disk_dbl, disk, disksize, cudaMemcpyDevicetoDevice);
		hipMemcpy(n_dbl, n, nsize, cudaMemcpyDevicetoDevice);
	}
	// Have fun! Grab a drink! 
	// changed a line
}

